#include "hip/hip_runtime.h"
extern "C" {
#include "train_gpu_cu.h"
  
}
#define N_UNARY 4
//TODO: copy V_data, to allocated V_data
extern "C" void GPU_grad_descent(gradient_t *args,i32 epochs,i32 dummy) {
  #define VERBOSE 0
  i32 h,i,j;

  gridCRF_t *self = args->self;
  npy_intp *dims=args->dims;
  i32 depth = self->depth;

  gpu_loopy_params_t lpar;
  lpar.max_its = args->lpar->max_its;
  lpar.stop_thresh = args->lpar->stop_thresh;
  lpar.eval = args->lpar->eval;

  i32 n_factors=args->n_factors;

  PyObject *X_list=args->X_list;
  PyObject *Y_list=args->Y_list;
  
  
  f32 totL;
  i32 n_samples = PyList_Size(X_list);
  hipError_t err = hipSuccess;

  
  f32 **mu_l = (f32 **) malloc(sizeof(f32*) * n_samples);
  i32 **EY_l = (i32 **) malloc(sizeof(i32*) * n_samples);

  f32 **X_l =  (f32 **) malloc(sizeof(f32*) * n_samples);
  i32 **Y_l =  (i32 **) malloc(sizeof(i32*) * n_samples);

  //cuda streams
  const i32 n_streams = 10;
  hipStream_t stream[n_streams];
  for (i=0;i<n_streams;i++) {
    hipStreamCreate(&stream[i]);
  }
  
  i32 curstream = 0;


  /* parameters */
  f32 * V_data;
  hipMalloc(&V_data, sizeof(f32)*(n_factors*8 + N_UNARY));//check this...
  err=hipMemcpyAsync(V_data, self->V_data, sizeof(f32)*(n_factors*8), hipMemcpyHostToDevice, stream[(curstream++)%n_streams]);
  assert(err==hipSuccess);
  
  f32 *unary_w = V_data + n_factors*8;//sizeof(f32)*n_factors*8;
  err=hipMemcpyAsync(unary_w, self->unary, N_UNARY*sizeof(f32), hipMemcpyHostToDevice, stream[(curstream++)%n_streams]);
  assert(err==hipSuccess);
  
  //TODO: copy V to V_data
  for (j=0;j<n_samples;j++){
    dims=PyArray_DIMS((PyArrayObject*)PyList_GetItem(X_list,j));
    //allocate space
    hipMalloc(&mu_l[j],dims[0]*dims[1]*2*sizeof(f32));
    
    hipMalloc(&EY_l[j],sizeof(i32)*dims[0]*dims[1]);

    hipMalloc(&X_l[j], sizeof(f32)*dims[0]*dims[1]*2);
    hipMalloc(&Y_l[j], sizeof(i32)*dims[0]*dims[1]*2);
    //Copy images to memory
    err=hipMemcpyAsync(X_l[j],PyArray_DATA(((PyArrayObject*) PyList_GetItem(X_list,j))), \
		    sizeof(f32)*dims[0]*dims[1]*2, hipMemcpyHostToDevice,
		    stream[(curstream++)%n_streams]);
    assert(err==hipSuccess);
    err=hipMemcpyAsync(Y_l[j], PyArray_DATA(((PyArrayObject*)PyList_GetItem(Y_list,j))), \
		    sizeof(i32)*dims[0]*dims[1]*2, hipMemcpyHostToDevice,\
		    stream[(curstream++)%n_streams]);
    assert(err==hipSuccess);
  }

  f32 *V_change;
  hipMalloc(&V_change, sizeof(f32)*(n_factors*4*2+N_UNARY));
  f32 *unary_change = V_change + n_factors*4*2;

  i32 **com_l = (i32**) malloc(sizeof(i32*)*n_samples);
  i32 **rom_l = (i32**) malloc(sizeof(i32*)*n_samples);
  om_pair **co_pairs_l = (om_pair**) malloc(sizeof(om_pair*)*n_samples);
  
  /* Prepare coordinates*/
  i32 *_com=(i32*) malloc(sizeof(i32)*n_factors);
  i32 *_rom=(i32*) malloc(sizeof(i32)*n_factors);
  om_pair *_co_pairs=(om_pair*)malloc(sizeof(om_pair)*n_factors);
  for (h=0;h<n_samples;h++) {
    dims=PyArray_DIMS((PyArrayObject*)PyList_GetItem(X_list,h));


    i32 n=0;
    for (j=1;j<=depth;j++ ) {
      for (i=0;i<j*4;i++) {
	if (i<j) {
	  _com[n]= -j *dims[1] * n_factors*2*2 - i*n_factors*2*2;
	  _rom[n]= +j *dims[1] * n_factors*2*2 + i*n_factors*2*2;
	  _co_pairs[n]=(om_pair){-j,-i};
	}
	else if (i>=j*3) {
	  _com[n]= +j *dims[1] * n_factors*2*2 - (j-(i-j*3))*n_factors*2*2;
	  _rom[n]= -j *dims[1] * n_factors*2*2 + (j-(i-j*3))*n_factors*2*2;
	  _co_pairs[n]=(om_pair){j,-(j-(i-j*3))};
	}
	else{
	  _com[n]= (-2*j+i)*dims[1] * n_factors*2*2 - j*n_factors*2*2;
	  _rom[n]= (2*j-i)*dims[1] * n_factors*2*2 + j*n_factors*2*2;
	  _co_pairs[n]=(om_pair){-2*j+i,-j};
	}
      
	n++;
      }
    }

    hipMalloc(&com_l[h],  sizeof(i32)*n_factors);
    err=hipMemcpyAsync(com_l[h], _com, sizeof(i32)*n_factors, hipMemcpyHostToDevice, stream[(curstream++)%n_streams]);
    assert(err==hipSuccess);
    hipMalloc(&rom_l[h],  sizeof(i32)*n_factors);
    err=hipMemcpyAsync(rom_l[h], _rom, sizeof(i32)*n_factors, hipMemcpyHostToDevice, stream[(curstream++)%n_streams]);
    assert(err==hipSuccess);
    hipMalloc(&co_pairs_l[h],  sizeof(om_pair)*n_factors);
    err=hipMemcpyAsync(co_pairs_l[h], _co_pairs, sizeof(om_pair)*n_factors, hipMemcpyHostToDevice, stream[(curstream++)%n_streams]);
    assert(err==hipSuccess);
    
  }
  /* End prepare coordinates*/

  

  f32 ** unary_c_l = (f32**) malloc(sizeof(f32*) * n_samples);
  for (j=0;j<n_samples;j++){
    dims=PyArray_DIMS((PyArrayObject*)PyList_GetItem(X_list,j));
    hipMalloc(&unary_c_l[j], dims[0]*dims[1]*2*sizeof(f32));
  }

  f32 *RE, *CE;
  hipMalloc(&RE, sizeof(f32) * 2* n_factors *2);
  hipMalloc(&CE, sizeof(f32) * 2* n_factors *2);

  f32 **V_F_l = (f32**) malloc(sizeof(f32*) * n_samples);
  f32 **F_V_l = (f32**) malloc(sizeof(f32*) * n_samples);

  for (j=0;j<n_samples;j++){
    dims=PyArray_DIMS((PyArrayObject*)PyList_GetItem(X_list,j));
    hipMalloc(&V_F_l[j], sizeof(f32)*dims[0]*dims[1]*n_factors*4);
    hipMalloc(&F_V_l[j], sizeof(f32)*dims[0]*dims[1]*n_factors*4);
  }


  i32 * ainc;
  i32 * binc;
  hipMalloc(&ainc, sizeof(i32)*n_factors*2);  
  hipMalloc(&binc, sizeof(i32)*n_factors*2);
  err=hipMemcpyAsync(ainc, args->ainc, sizeof(i32)*n_factors*2, hipMemcpyHostToDevice, stream[(curstream++)%n_streams]);
  assert(err==hipSuccess);
  err=hipMemcpyAsync(binc, args->binc, sizeof(i32)*n_factors*2, hipMemcpyHostToDevice, stream[(curstream++)%n_streams]);
  assert(err==hipSuccess);
  
  
  
  gpu_gradient_t g_args;
  g_args.self = self;
  g_args.dev_ainc = ainc;
  g_args.dev_binc = binc;
  g_args.num_params= args->num_params;
  g_args.n_factors = args->n_factors;
  g_args.alpha=args->alpha;
  g_args.lpar = &lpar;
  g_args.host_L = 0.0;
  g_args.dev_V_change = V_change;
  g_args.dev_unary_change = unary_change;
  
  hipMalloc(&(g_args.dev_L),sizeof(f32));
  
  gpu_loopy_data gdata;
  gdata.V_data = V_data;
  gdata.RE = RE;
  gdata.CE = CE;
  gdata.unary_w = unary_w;
  
  g_args.gdata = &gdata;
  lpar.gdata = &gdata;

  for (i=0;i<n_streams;i++) {
    hipStreamDestroy(stream[i]);
  }

  //shuffle the training examples
  srand(0);
  i32 *inds = indlist(n_samples);

  
  for (i=0;i < epochs;i++) {
    shuffle_inds(inds, n_samples);
    for (j=0;j < n_samples;j++){
      dims=PyArray_DIMS((PyArrayObject*)PyList_GetItem(X_list,inds[j]));
      
      gdata.V_F = V_F_l[inds[j]];
      gdata.F_V = F_V_l[inds[j]];
      gdata.mu = mu_l[inds[j]];
      gdata.com = com_l[inds[j]];
      gdata.rom = rom_l[inds[j]];
      gdata.co_pairs = co_pairs_l[inds[j]];
      gdata.unary_c = unary_c_l[inds[j]];
      gdata.EY = EY_l[inds[j]];
      gdata.X = X_l[inds[j]];

      g_args.dev_X = X_l[inds[j]];
      g_args.dev_Y = Y_l[inds[j]];
      g_args.dims= dims;

      loopyGPU(self, (PyArrayObject*)PyList_GetItem(X_list,inds[j]), &lpar, NULL);
      gpu_calculate_gradient(&g_args);
    }
  }
  free(inds);

  
  // copy V_data back to numpy space...
  // also copy unary data back to numpy space3
  hipMemcpy(self->V_data, V_data,  sizeof(f32)*(n_factors*8), hipMemcpyDeviceToHost);
  hipMemcpy(self->unary, unary_w,  sizeof(f32)*(N_UNARY), hipMemcpyDeviceToHost);

  //Time to clean up everything
  hipFree(V_data);
  hipFree(RE);
  hipFree(CE);

  

  hipFree(g_args.dev_L);
  for (j=0;j<n_samples;j++){
    hipFree(mu_l[j]);
    hipFree(EY_l[j]);
    hipFree(X_l[j]);
    hipFree(Y_l[j]);
    hipFree(com_l[j]);
    hipFree(rom_l[j]);
    hipFree(co_pairs_l[j]);
    hipFree(unary_c_l[j]);
    hipFree(V_F_l[j]);
    hipFree(F_V_l[j]);
  }
  hipFree(V_change);
  hipFree(ainc);
  hipFree(binc);
  
  free(mu_l);
  free(EY_l);
  free(X_l);
  free(Y_l);
  free(_com);
  free(_rom);
  free(_co_pairs);
}



static void gpu_calculate_gradient(gpu_gradient_t *args) {
  f32 *X = args->dev_X;
  i32 *Y = args->dev_Y;
  i32 *ainc = args->dev_ainc, *binc = args->dev_binc;
  f32 *V_change = args->dev_V_change;
  f32 *unary_change = args->dev_unary_change;
  f32 *unary_w = args->gdata->unary_w;
  f32 *unary_c = args->gdata->unary_c;
  
  f32 *L = args->dev_L;
  npy_intp *dims = args->dims;
  i32 n_factors = args->self->n_factors;

  i32 * EY = args->gdata->EY;
  f32 * V = args->gdata->V_data;

  hipStream_t stream;
  hipStreamCreate(&stream);
  
  i32 n_elem;
  n_elem=(n_factors*4*2+N_UNARY);
  dim3 blockGrid(n_elem/128 + 1);
  dim3 threadGrid(128);
  gpu_fill_value<<<blockGrid, threadGrid,0,stream>>>(V_change,0.0, n_elem);
  //note, this also fills unary_change

  f32 alpha = args->alpha;
  dim3 factorgrid(2*n_factors,2);
  dim3 singGrid(2);
  gpu_loopy_V_F__computeunary<<<factorgrid, singGrid,0 ,stream >>>(X, unary_w, unary_c);

  dim3 blockGrid1(dims[0]/16 + 1, dims[1]/16 + 1);
  dim3 threadGrid1(16,16,2);

  
  gpu_entropy_partial<<<blockGrid1, threadGrid1, sizeof(f32)*16*16*2, stream >>> (unary_c, EY, X, Y, V, V_change, unary_change, ainc, binc, alpha, (i32) dims[0], (i32) dims[1], n_factors);
  
  dim3 blockGrid2(1);
  dim3 threadGrid2(n_factors*8 + N_UNARY);
  gpu_update_params<<<blockGrid2, threadGrid2,0 , stream>>> (V, V_change, 1.0/(dims[0]*dims[1])); //this also includes unary changes

  hipStreamDestroy(stream);
}


__global__ void gpu_entropy_partial(f32 *unary_c, i32 *EY, f32 *X, i32 *Y, f32 *V, f32 *V_change, f32* unary_change, i32 *ainc, i32 *binc, f32 alpha, i32 limx, i32 limy, i32 n_factors) {

  //last pitch idea. Forget cond until the very end
  
  // TODO: optimize by putting everything V into shared data.
  // Also, possibly EY

  //TODO: check page 83, mentions that V_change and unary need to be properly aligned.
  i32 x = blockIdx.x * 16 + threadIdx.x;
  i32 y = blockIdx.y * 16 + threadIdx.y;
  i32 c= threadIdx.z;
  i32 i;
  i32 l;
  i32 co = ((x)*limy + y);
  i32 cond= (x >= limx || y >= limy) || (Y[co*2+c]==0 && Y[co*2+c^1]==0);
  extern __shared__ char array[];
  //f32 *shared_V = (f32*) array;  // can copy this by using elements in reange

  f32 *shared_sum = (f32*) array ;//+ n_factors*8*sizeof(f32);
  f32 sum, max, s1, change;
  __syncthreads();
  
  if (!cond) {
    sum = -unary_c[2*co+c];
    
    for (i=0;i<n_factors;i++) {
      if (x+ainc[i] < 0 || x+ainc[i]>=limx || y+binc[i] < 0 || y+binc[i] >= limy) continue;
      l= EY[COORD2(x+ainc[i],y+binc[i],limx,limy,1)];
      sum += V[i*4 + (l)*2 + c];
    }
    for (i=0;i<n_factors;i++) {
      if (x+ainc[i+n_factors] < 0 || x+ainc[i+n_factors]>=limx || y+binc[i+n_factors] < 0 || y+binc[i+n_factors] >= limy) continue;
    
      l= EY[COORD2(x+ainc[i+n_factors],y+binc[i+n_factors],limx,limy,1)];
      sum += V[n_factors*4 + i*4 + (l)*2 + c];
    }
    
    //put sum into shared memory
    shared_sum[threadIdx.x*16*2 + threadIdx.y*2 +c] = sum;
  }
  __syncthreads();
  if(!cond) {
    
    if (sum < shared_sum[threadIdx.x*16*2 + threadIdx.y*2+c^1]){
      max = sum;
    }
    else{
      max = shared_sum[threadIdx.x*16*2 + threadIdx.y*2+c^1];
    }
  }
  __syncthreads();
  if (!cond) {  
    s1 = expf(-shared_sum[threadIdx.x*16*2 + threadIdx.y*2+c]-max);

    shared_sum[threadIdx.x*16*2 + threadIdx.y*2+c] = s1;
  }
  __syncthreads();
  // Each thread handles the specific class
  //Softmax
  if (!cond) {
    l = Y[co*2+c];
    s1= shared_sum[threadIdx.x*16*2 + threadIdx.y*2+c] / (shared_sum[threadIdx.x*16*2 + threadIdx.y*2]+shared_sum[threadIdx.x*16*2 + threadIdx.y*2+1]); 

    
    change = -alpha*(l-s1);
    //printf("%d %d %d %f %d %f\n", threadIdx.x, threadIdx.y, c, s1, l, change);


    atomicAdd(&unary_change[c*2], change*X[co*2]);
    atomicAdd(&unary_change[c*2+1], change*X[co*2+1]);
    
    //possible optimization
    for (i=0;i<n_factors;i++) {
      if (x+ainc[i] < 0 || x+ainc[i]>=limx || y+binc[i] < 0 || y+binc[i] >= limy) continue;
      l= EY[COORD2(x+ainc[i],y+binc[i],limx,limy,1)];
    //Atomic add
      atomicAdd(&V_change[i*4 + 2*l +c], change);
    }
    
    for (i=0;i<n_factors;i++) {
      if (x+ainc[n_factors+i] < 0 || x+ainc[n_factors+i]>=limx || y+binc[n_factors+i] < 0 || y+binc[n_factors+i] >= limy) continue;
      
      l= EY[COORD2(x+ainc[n_factors+i],y+binc[n_factors+i],limx,limy,1)];
      //Atomic add
      atomicAdd(&V_change[n_factors*4 + i*4 + 2*l +c], change);
    }
  }
}

__global__ void gpu_update_params(f32 *V, f32* V_change, f32 lr) {
  V[threadIdx.x] += lr*V_change[threadIdx.x];
}

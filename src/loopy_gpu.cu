#include "hip/hip_runtime.h"
extern "C" {
#include "loopy_gpu_cu.h"
}

//#define CPU_TEST

#ifdef CPU_TEST
void __debug_loopy_V_F(loopygpu_t *targs);
void *_loopy_label(loopygpu_t *l_args);
#endif
  
/* 
Optimization strats:
1. Split F_V so that it can take advantage of constant memory for offset coordinates. This can speed up execution possibly.
I.e.
have one block take care of factor index 1 for a portion of variables.

*/

extern "C" i32 *loopyGPU(gridCRF_t* self, PyArrayObject *X_py,loopy_params_t *lpar,PyArrayObject *refimg){
  npy_intp * dims= PyArray_DIMS(X_py);
  i64 n_factors=self->n_factors;
  i64 max_it=lpar->max_its,it;

  f32 * V_data=self->V_data;
  i64 n,depth=self->depth,i,j;

  
  f32 * _F_V = (f32 *) _mm_malloc( dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32),32);
  f32 * _V_F = (f32 *) _mm_malloc( dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32),32);
  
  for (i=0;i<dims[0] * dims[1] * (n_factors*2) *2; i++){
    _F_V[i]=0.0f;
    _V_F[i]=0.0f;
  }


  f32 *_mu = lpar->mu;
  f32 * _unary_w = self->unary;
  
  for (i=0;i<dims[0]*dims[1]*2;i+=1) {
    _mu[i]=BIG;

  }
  

  /* Prepare coordinates*/
  i32 *_com=(i32*)malloc(sizeof(i32)*n_factors);
  i32 *_rom=(i32*)malloc(sizeof(i32)*n_factors);
  om_pair *_co_pairs=(om_pair*)malloc(sizeof(om_pair)*n_factors);

  n=0;
  for (j=1;j<=depth;j++ ) {
    for (i=0;i<j*4;i++) {
      if (i<j) {
	_com[n]= -j *dims[1] * n_factors*2*2 - i*n_factors*2*2;
	_rom[n]= +j *dims[1] * n_factors*2*2 + i*n_factors*2*2;
	_co_pairs[n]=(om_pair){-j,-i};
      }
      else if (i>=j*3) {
	_com[n]= +j *dims[1] * n_factors*2*2 - (j-(i-j*3))*n_factors*2*2;
	_rom[n]= -j *dims[1] * n_factors*2*2 + (j-(i-j*3))*n_factors*2*2;
	_co_pairs[n]=(om_pair){j,-(j-(i-j*3))};
      }
      else{
	_com[n]= (-2*j+i)*dims[1] * n_factors*2*2 - j*n_factors*2*2;
	_rom[n]= (2*j-i)*dims[1] * n_factors*2*2 + j*n_factors*2*2;
	_co_pairs[n]=(om_pair){-2*j+i,-j};
      }
      
      n++;
    }
  }
  
  n=0;


  /* transfer matrices */
  f32 *_RE= (f32 *) _mm_malloc(2 * n_factors * 2 * sizeof(f32),32); 
  f32 *_CE= (f32 *) _mm_malloc(2 * n_factors * 2 * sizeof(f32),32); 
  __m256 r1,r2;


  for (i=0;i<2*n_factors*2;i+=8) {
    r1=_mm256_load_ps(&V_data[i]);
    r2=_mm256_load_ps(&V_data[i + n_factors*4]);
    //r1=exp256_ps(r1);
    //assert (!(isnan(r1[6]) || isnan(r1[7])));

    /*Swap energies such that remote outcome=1 is seperated from
      remote outcome=0*/
    _RE[i/2]=-r1[0];
    _RE[i/2+1]=-r1[1];
    _RE[n_factors*2+i/2]=-r1[2];
    _RE[n_factors*2+i/2+1]=-r1[3];
    _RE[i/2+2]=-r1[4];
    _RE[i/2+3]=-r1[5];
    _RE[n_factors*2+i/2+2]=-r1[6];
    _RE[n_factors*2+i/2+3]=-r1[7];
    
    
    _CE[i/2]=-r2[0];
    _CE[i/2+1]=-r2[1];
    _CE[n_factors*2+i/2]=-r2[2];
    _CE[n_factors*2+i/2+1]=-r2[3];
    _CE[i/2+2]=-r2[4];
    _CE[i/2+3]=-r2[5];
    _CE[n_factors*2+i/2+2]=-r2[6];
    _CE[n_factors*2+i/2+3]=-r2[7];
  
  }
 
  loopygpu_t targs;
  // set up threads

  const i32 n_streams = 10;
  hipStream_t stream[n_streams];
  for (i=0;i<n_streams;i++) {
    hipStreamCreate(&stream[i]);
  }
  
  /* Allocate message buffers*/
  f32 *F_V, *V_F;
  hipMalloc(&F_V, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32));
  hipMalloc(&V_F, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32));

  hipMemcpyAsync(F_V, _F_V, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32), hipMemcpyHostToDevice, stream[0]);
hipMemcpyAsync(V_F,_V_F, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32), hipMemcpyHostToDevice, stream[1]);

  /* Allocate coordinate system*/
  i32 *com, *rom;
  om_pair *co_pairs;
  hipMalloc(&com, sizeof(i32)*n_factors);
  hipMalloc(&rom, sizeof(i32)*n_factors);
  hipMalloc(&co_pairs,sizeof(om_pair)*n_factors);

  hipMemcpyAsync(com, _com, n_factors* sizeof(i32), hipMemcpyHostToDevice, stream[2]);
  hipMemcpyAsync(rom, _rom, n_factors* sizeof(i32), hipMemcpyHostToDevice, stream[3]);
  hipMemcpyAsync(co_pairs, _co_pairs, n_factors* sizeof(om_pair), hipMemcpyHostToDevice, stream[4]);

  f32 *RE, *CE;
  hipMalloc(&RE, 2* n_factors *2* sizeof(f32));
  hipMalloc(&CE, 2* n_factors *2* sizeof(f32));
  hipMemcpyAsync(RE, _RE, 2*n_factors*2*sizeof(f32), hipMemcpyHostToDevice, stream[5]);
  hipMemcpyAsync(CE, _CE, 2*n_factors*2*sizeof(f32), hipMemcpyHostToDevice, stream[6]);

  f32 *unary_w, *unary_c;
  hipMalloc(&unary_w, 4 * sizeof(f32));
  hipMemcpyAsync(unary_w, _unary_w, 4*sizeof(f32), hipMemcpyHostToDevice, stream[7]);
    
  hipMalloc(&unary_c, dims[0]*dims[1]*2*sizeof(f32));

  
  f32 *mu;
  hipMalloc(&mu, dims[0]*dims[1]*2*sizeof(f32));
  hipMemcpyAsync(mu, _mu, dims[0]*dims[1]*2*sizeof(f32), hipMemcpyHostToDevice, stream[8]);

  f32 *X;
  hipMalloc(&X, dims[0]*dims[1]*2*sizeof(f32));//tmp
  hipMemcpyAsync(X, X_py->data, dims[0]*dims[1]*2*sizeof(f32), hipMemcpyHostToDevice, stream[9]);

  for (i=0;i<n_streams;i++) {
    hipStreamSynchronize(stream[i]);
    hipStreamDestroy(stream[i]);
  }
  
  i32 *converged;
  hipMalloc(&converged,sizeof(i32));
  i32 _converged = 1;

  targs.com=com;
  targs.rom=rom;
  targs.co_pairs = co_pairs;
  targs.X=X;
  targs.refimg=NULL;
  targs.lpar = lpar;
  targs.self = self;
  targs.F_V = F_V;
  targs.V_F = V_F;
  targs.RE = RE;
  targs.CE = CE;
  targs.mu = mu;
  targs.unary_w=unary_w;
  targs.unary_c=unary_c;
  targs.X_py = X_py;
  targs.converged = converged;
  targs._converged = &_converged;


  //DELETE THIS
  targs._F_V=_F_V;
  targs._V_F=_V_F;
  targs._mu=_mu;
  targs._unary=_unary_w;
  //
  for (it = 0; it < max_it; it++){
    if (it%10==0){
      printf("gpu it %d\n", it);
    }
    gpu_loopy_F_V(&targs);
    #ifndef CPU_TEST
    gpu_loopy_V_F(&targs);
    #else
    __debug_loopy_V_F(&targs);
    #endif
    
    if (_converged) break;
  }
  printf("converged %d %f\n",_converged, lpar->stop_thresh);

  #ifndef CPU_TEST
  i32 *EY;
  hipMalloc(&EY, dims[0]*dims[1]*sizeof(f32));
  
  dim3 dimGrid(dims[0],dims[1]);
  dim3 singGrid(2);
  gpu_loopy_V_F__label<<<dimGrid, singGrid, 2*sizeof(f32)>>>(F_V, unary_c, EY, n_factors);

  hipMemcpy(lpar->EY, EY, dims[0]*dims[1]*sizeof(i32), hipMemcpyDeviceToHost);
  #else
  _loopy_label(&targs);
  #endif
  
  
 cleanup:
  _mm_free(_F_V);
  _mm_free(_V_F);
  free(_com);
  free(_rom);
  free(_co_pairs);
  _mm_free(_RE);
  _mm_free(_CE);

  hipFree(F_V);
  hipFree(V_F);
  hipFree(com);
  hipFree(rom);
  hipFree(co_pairs);
  hipFree(RE);
  hipFree(CE);
  hipFree(unary_w);
  hipFree(unary_c);
  hipFree(mu);
  hipFree(X);
  hipFree(converged);
#ifndef CPU_TEST
  hipFree(EY);
#endif

  return lpar->EY;
}


extern "C" void gpu_loopy_F_V(loopygpu_t *targs) { 

  npy_intp * dims= PyArray_DIMS(targs->X_py);
  gridCRF_t *self = targs->self;
  
  i32 n_factors=self->n_factors;
  
  f32 *F_V= targs->F_V;
  f32 *V_F= targs->V_F;

  f32 *RE = targs->RE;
  f32 *CE = targs->CE;

  i32 *com = targs->com;
  i32 *rom = targs->rom;
  om_pair * co_pairs = targs->co_pairs;
  
  i32 i;

  const i32 n_streams = 2;
  hipStream_t stream[n_streams];
  for (i=0;i<n_streams;i++) {
    hipStreamCreate(&stream[i]);
  }
  

  dim3 dimGrid(dims[0],dims[1]);
  dim3 factorgrid(n_factors);
		  
  gpu_loopy_F_V__Flow<<<dimGrid, factorgrid, 0, stream[0]>>>(F_V, V_F, RE, NULL, rom, co_pairs, n_factors);
  gpu_loopy_F_V__Fup<<<dimGrid, factorgrid, 0, stream[1]>>>(F_V, V_F, CE, NULL, com, co_pairs, n_factors);

  for (i=0;i<2;i++) {
    hipStreamDestroy(stream[i]);
  }

}


/* Naive method */
__global__ void gpu_loopy_F_V__Flow(f32 *F_V, f32 *V_F, f32 *RE, const i32 * refimg, const i32 * com, const om_pair * co_pairs,  i32 n_factors){
  /* Naive code*/
  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 n = threadIdx.x;
  //Note, may need to swap gridDim.x and gridDim.y

  /* Check bounds for upper factor */
  om_pair cop = co_pairs[n];
  if ( ! (x+cop.x <0 || x+cop.x >= gridDim.x || y+cop.y < 0 || y+cop.y >=gridDim.y) ){//&& !(refimg[COORD2(x+cop.x,y+cop.y, gridDim.x, gridDim.y, 1)]==0)) {
    i32 origin=COORD3(x,y,0,gridDim.x,gridDim.y,2*n_factors,2);
    i32 co = origin + com[n] + 2*(n + n_factors);
  
    F_V[co] = RE[n*2] + V_F[origin] > RE[n_factors*2 + n*2] + V_F[origin+1] ? RE[n*2] + V_F[origin] : RE[n_factors*2 + n*2] + V_F[origin+1];
  
    F_V[co+1] = RE[n*2+1] + V_F[origin] > RE[n_factors*2 + n*2 + 1] + V_F[origin+1] ? RE[n*2+1] + V_F[origin] : RE[n_factors*2 + n*2 + 1] + V_F[origin+1];
  }
}

__global__ void gpu_loopy_F_V__Fup(f32 *F_V, f32 *V_F,  f32 *CE, const i32 * refimg, const i32 *rom, const om_pair *co_pairs, i32 n_factors){
  /* Naive code*/
  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 n = threadIdx.x;
  

  /* Check bounds for upper factor */
  om_pair cop=co_pairs[n];
  /* Check bounds for lower factor */
  if (!(x-cop.x < 0 || x-cop.x >= gridDim.x || y-cop.y < 0 || y-cop.y >=gridDim.y)){// && !(refimg[COORD2(x-cop.x,y-cop.y, gridDim.x, gridDim.y, 1)]==0)) {
    i32 origin=COORD3(x,y,0,gridDim.x,gridDim.y,2*n_factors,2);
    //i32 co=origin + rom[n]; //check this
    i32 co = origin+rom[n] + 2*n;
    F_V[co] = CE[n*2] + V_F[origin] > CE[n_factors*2 + n*2] + V_F[origin+1] ?
      CE[n*2] + V_F[origin] : CE[n_factors*2 + n*2] + V_F[origin+1];
  
    F_V[co+1] = CE[n*2+1] + V_F[origin] > CE[n_factors*2 + n*2 + 1] + V_F[origin+1] ?
      CE[n*2+1] + V_F[origin] : CE[n_factors*2 + n*2 + 1] + V_F[origin+1];
  }
  
}



extern "C" void gpu_loopy_V_F(loopygpu_t *targs) {


  //TODO:
  // - copy X to gpu

  i32 i;
  gridCRF_t *self = targs->self;
  f32 *X = targs->X;

  loopy_params_t * lpar = targs->lpar;
  
  npy_intp * dims= PyArray_DIMS(targs->X_py);
  i64 n_factors=self->n_factors;
  f32 stop_thresh=lpar->stop_thresh;
  

  f32 * unary_w= targs->unary_w;
  f32 * unary_c = targs->unary_c;

  f32 *F_V = targs->F_V;
  f32 *V_F = targs->V_F;
  f32 *mu = targs->mu;

  /* runtime Flags*/
  i32 *converged = targs->converged;
  
  const i32 n_streams = 1;
  hipStream_t stream[n_streams];
  for (i=0;i<n_streams;i++) {
    hipStreamCreate(&stream[i]);
  }
  
  i32 tmp=1;
  hipMemcpyAsync(converged, &tmp, sizeof(i32),hipMemcpyHostToDevice,stream[0]);

  for (i=0;i<n_streams;i++) {
    hipStreamSynchronize(stream[i]);
  }
//#define DEBUG
  #ifdef DEBUG
  f32 *oldmu = (f32*) malloc(sizeof(f32)*dims[0]*dims[1]*2);
  f32 *newmu = (f32*) malloc(sizeof(f32)*dims[0]*dims[1]*2);
  hipMemcpy(oldmu, mu, sizeof(f32)*dims[0]*dims[1]*2, hipMemcpyDeviceToHost);
  #endif
  
  dim3 dimGrid(dims[0],dims[1]);
  dim3 factorgrid(2*n_factors,2);
  dim3 singGrid(2);
  gpu_loopy_V_F__computeunary<<<dimGrid, singGrid, 0, stream[0]>>>(X, unary_w, unary_c);
  gpu_loopy_V_F__sumfactors<<<dimGrid, factorgrid, sizeof(f32)*n_factors*8, stream[0]>>>(F_V, V_F, unary_c, NULL, n_factors);
  gpu_loopy_V_F__marginal<<<dimGrid, singGrid, 0, stream[0]>>>(F_V, unary_c, mu, n_factors, stop_thresh, converged);

  for (i=0;i<n_streams;i++) {
    hipStreamDestroy(stream[i]);
  }

  #ifdef DEBUG
  i32 rip =0;
  hipMemcpy(newmu, mu, sizeof(f32)*dims[0]*dims[1]*2, hipMemcpyDeviceToHost);
  for (i=0;i<dims[0]*dims[1]*2 && !rip;i++) {
    if (fabsf(oldmu[i]-newmu[i]) > stop_thresh){
      printf("WTF");
      rip=1;
    }
    if (isnan(oldmu[i]) || isnan(newmu[i])){
      printf("We have a nan\n");
      rip=1;
    }
    //printf("%f ",oldmu[i]-newmu[i]);
  }
  if (rip) {
    printf("\n\n");
  }

  #endif
  hipMemcpy(targs->_converged, converged, sizeof(i32), hipMemcpyDeviceToHost);
 
}

__global__ void gpu_loopy_V_F__computeunary(f32 * X, f32 *unary_w, f32 *unary_c){
    // one possibility. X is not aligned properly.
  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 c = threadIdx.x;

  unary_c[COORD2(x,y,gridDim.x, gridDim.y, 2) + c] = -(	    \
    X[COORD2(x,y,gridDim.x, gridDim.y, 2)] * unary_w[c*2] + \
      X[COORD2(x,y,gridDim.x, gridDim.y, 2) + 1] * unary_w[c*2 + 1]);
    
}

__global__ void gpu_loopy_V_F__sumfactors(f32 *F_V, f32 *V_F, f32 *unary_c, const i32 * refimg, i32 n_factors ){
  extern __shared__ char array[];
  f32 *shared_f_v = (f32*) array;
  f32 *shared_v_f = (f32*) (array + sizeof(f32)*n_factors*2*2);

  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 n = threadIdx.x;
  i32 c = threadIdx.y;
  i32 i=0;
  i32 origin = COORD3(x,y,n,gridDim.x, gridDim.y, 2*n_factors, 2) + c;
  // load factor to vvariables into shared memory
  shared_f_v[n*2 + c] = F_V[origin];

  //TODO: make unary a constant?
  f32 sum = unary_c[COORD2(x,y,gridDim.x,gridDim.y,2) + c] - shared_f_v[n*2+c];
  __syncthreads();


  /* Sum up all messages */
  for (i=0;i<2*n_factors;i++) {
//printf("KEK %d\n", i*2+c);
    sum += shared_f_v[i*2+c]; //this is the problem!
  }
  shared_v_f[n*2+c]=sum;
  
  __syncthreads();  
  // Normalize values
  sum = sum - 0.5 * (sum+shared_v_f[n*2+c^1]);
  V_F[origin]=sum;
  
}

/* TODO: finish */
__global__ void gpu_loopy_V_F__marginal(f32 *F_V, f32 * unary_c,  f32 * mu, i32 n_factors, f32 stop_thresh, i32 *converged) {
  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 c = threadIdx.x;

  i32 i;
  i32 origin = COORD2(x,y,gridDim.x, gridDim.y, 2) + c;
  f32 sum = unary_c[origin];

  // sum up factors
  for (i=0;i<n_factors*2;i++) {
    sum += F_V[COORD3(x,y,i,gridDim.x, gridDim.y, 2*n_factors, 2) + c];
  }

  if (fabsf(sum - mu[origin]) > stop_thresh) {
    converged[0] = 0;
  }
  mu[origin] = sum;
}

//TODO: change to mu instead
__global__ void gpu_loopy_V_F__label(f32 *F_V, f32 * unary_c, i32 *EY, i32 n_factors) {

  /* Computes the predicted label given the values */
  extern __shared__ char array[];
  f32 *shared_marginal = (f32*) array;
  i32 i;
  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 c = threadIdx.x;
  i32 origin = COORD2(x,y,gridDim.x, gridDim.y, 2) + c;

  f32 sum = unary_c[origin];
  // sum up factors
  for (i=0;i<n_factors*2;i++) {
    sum += F_V[COORD3(x,y,i,gridDim.x, gridDim.y, 2*n_factors, 2) + c];
  }
  shared_marginal[c] = sum;
  __syncthreads();
  if (c==0  && shared_marginal[0] > shared_marginal[1]) {
    EY[COORD2(x,y,gridDim.x, gridDim.y, 1)] = 0;
  }
  else if (c==0) {
    EY[COORD2(x,y,gridDim.x, gridDim.y, 1)] = 1;
  }
  
}





#ifdef CPU_TEST



void __debug_loopy_V_F(loopygpu_t *targs) {

  /* Compute factor to variable messages */
  i32 WARN_FLAG=1;
  i32 i,j;
  gridCRF_t *self = targs->self;
  PyArrayObject *X = targs->X_py;
  loopy_params_t * lpar = targs->lpar;
  
  f32 a,b;
  npy_intp * dims= PyArray_DIMS(X);
  i64 n_factors=self->n_factors;
  f32 stop_thresh=lpar->stop_thresh;
  
  f32 * unary = self->unary;
  
  npy_intp x,y;
  i32 n;


  f32 *F_V = targs->_F_V;
  f32 *V_F = targs->_V_F;

  f32 *gF_V = targs->F_V;
  f32 *gV_F = targs->V_F;

  hipMemcpy(F_V, gF_V, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32), hipMemcpyDeviceToHost);
  hipMemcpy(V_F, gV_F, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32), hipMemcpyDeviceToHost);
  
  
  f32 *marginals = (f32*) _mm_malloc(sizeof(f32)*dims[0]*dims[1]*2,32);
  f32 *mu = targs->_mu;

  /* coordinates */

  /* runtime Flags*/
  i32 *converged = targs->_converged;
  
  i32 origin;

  f32 tmp[2];

  __m256 r1,r2;
  /* Compute variable to factor messages */
  for (x=0;x<dims[0];x++) {
    for (y=0;y<dims[1];y++) {
      //variable to factor messages
      
      f64 base= *((f64*)PyArray_GETPTR3(X,x,y,0));
      *((f64*)tmp) = base;
      tmp[0]=-(((f32*)&base)[0]*unary[0] + ((f32*)&base)[1]*unary[1]);
      tmp[1]=-(((f32*)&base)[0]*unary[2] + ((f32*)&base)[1]*unary[3]);
      r1=(__m256)_mm256_set1_pd(*((f64*)tmp)); //set all elements in vector this thi
      //Warning: possible segfault
	
      for (n=0;n<n_factors*2;n+=4) { //Set baseline, since we know that unary is added to each V_F
	_mm256_store_ps(&V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)] ,r1);
      }
      
      for (i=0;i<n_factors*2;i++) {
	base=*((f64*)(&F_V[COORD3(x,y,i,dims[0],dims[1],2*n_factors,2)]));
	r1=(__m256)_mm256_set1_pd(base);
	for (n=0;n<n_factors*2;n+=4) {
	  r2=_mm256_load_ps(&V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)]);
	  r2=_mm256_add_ps(r2,r1);
	  _mm256_store_ps(&V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)],r2);
	}
      }
      for (n=0;n<n_factors*2;n+=8) { //correct double counting
	r1=_mm256_load_ps(&F_V[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)]);
	r2=_mm256_load_ps(&V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)]);
	r2=_mm256_sub_ps(r2,r1);
	_mm256_store_ps(&V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)],r2);
      }

      /*
      //TODO: normalize
      //This is SSE normalization. Unless more knowledge is gained, it would be slower to use these than individually going over each value.
      */
	
      //Apply normalization
      for (n=0;n<n_factors*2;n++) {
	//TODO: optimize
	a=V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)];
	b=V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)+1];
	a=0.5*(a+b);
	V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)]-=a;
	V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)+1]-=a;
      }
	
      //TODO:Add to marginals
      origin=COORD2(x,y,dims[0],dims[1],2);

      assert (origin < dims[0]*dims[1]*2);

      marginals[origin]=tmp[0];
      marginals[origin+1]=tmp[1];
      for (i=0;i<n_factors*2;i++) {

	assert(COORD3(x,y,i,dims[0],dims[1],2*n_factors,2)+ 1 < dims[0] * dims[1] * (n_factors*2) *2 && COORD3(x,y,n,dims[0],dims[1],2*n_factors,2) > 0);
	marginals[origin]+=F_V[COORD3(x,y,i,dims[0],dims[1],2*n_factors,2)]; // invalid read of 4
	marginals[origin+1]+=F_V[COORD3(x,y,i,dims[0],dims[1],2*n_factors,2)+1];
	if (isnan(marginals[origin]) && WARN_FLAG) {
	  printf("MARGINAL WARNING %d %d\n",x,y);
	  for (j=0;j<n_factors*2;j++) {
	    printf("%f %f\n",F_V[COORD3(x,y,j,dims[0],dims[1],2*n_factors,2)],F_V[COORD3(x,y,j,dims[0],dims[1],2*n_factors,2)+1]); // invalid read of 4
	  }
	  WARN_FLAG=0;
	}
      }
      a=fabs(marginals[origin]-mu[origin]);
      if (a > stop_thresh) {
	*converged = 0;
      }
      a=fabs(marginals[origin+1]-mu[origin+1]);
      if (a > stop_thresh) {
	*converged = 0;
      }

      mu[origin]=marginals[origin];
      mu[origin+1]=marginals[origin+1];
      //TODO: calculate marginal
    }
  }

  _mm_free(marginals);
  hipMemcpy(gF_V, F_V, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32), hipMemcpyHostToDevice);
  hipMemcpy(gV_F, V_F, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32), hipMemcpyHostToDevice);
}


void *_loopy_label(loopygpu_t *l_args) {
  loopy_params_t * lpar = l_args->lpar;
  npy_intp * dims= PyArray_DIMS(l_args->X_py);
  i32 *ret=lpar->EY;
  f32 *mu=l_args->_mu;
  i32 x,y;
  i32 origin;
  for (x=0;x<dims[0];x++) {
    for (y=0;y<dims[1];y++) {
   
      origin=COORD2(x,y,dims[0],dims[1],2); 
      assert(origin >= 0 && origin + 1 < dims[0]*dims[1]*2);
      if (mu[origin] > mu[origin+1]) {
	ret[COORD2(x,y,dims[0],dims[1],1)]=0;
      }
      else{
	ret[COORD2(x,y,dims[0],dims[1],1)]=1;

      }
    }

  }

  //hipMemcpy(ret, gEY, dims[0] * dims[1] * sizeof(i32), hipMemcpyHostToDevice);

  return NULL;

}
#endif

#include "hip/hip_runtime.h"
extern "C" {
#include "loopy_gpu_cu.h"
}

//#define CPU_TEST

#ifdef CPU_TEST
void __debug_loopy_V_F(loopygpu_t *targs);
void *_loopy_label(loopygpu_t *l_args);
#endif
  
/* 
Optimization strats:
1. Split F_V so that it can take advantage of constant memory for offset coordinates. This can speed up execution possibly.
I.e.
have one block take care of factor index 1 for a portion of variables.

*/

extern "C" i32 *loopyGPU(gridCRF_t* self, PyArrayObject *X_py, gpu_loopy_params_t *lpar, PyArrayObject *refimg){
  npy_intp * dims= PyArray_DIMS(X_py);
  i64 n_factors=self->n_factors;
  i64 max_it=lpar->max_its,it;

  
  f32 *V_data = lpar->V_data;
  i64 n,depth=self->depth,i,j;


  /* transfer matrices */
  loopygpu_t targs;
  // set up threads

  const i32 n_streams = 10;
  hipStream_t stream[n_streams];
  for (i=0;i<n_streams;i++) {
    hipStreamCreate(&stream[i]);
  }
  
  /* Allocate message buffers*/
  f32 *F_V = lpar->F_V, *V_F = lpar->V_F;
  n_elem = dims[0] * dims[1] * (n_factors*2) *2;
  dim3 blockGrid(n_elem/128 + 1);
  dim3 threadGrid(128);
  gpu_fill_value<<<blockGrid, threadgrid,0, stream[(curstream++)%n_streams]>>>(F_V, 0.0, n_elem);
  gpu_fill_value<<<blockGrid, threadgrid,0, stream[(curstream++)%n_streams]>>>(V_F, 0.0, n_elem);
  
  /* Allocate coordinate system*/
  i32 *com = lpar->com, *rom = lpar->rom;
  om_pair *co_pairs = lpar->co_pairs;

  f32 *mu = lpar->mu;
  n_elem = dims[0] * dims[1] *2;
  dim3 blockGrid(n_elem/128 + 1);
  dim3 threadGrid(128);
  gpu_fill_value<<<blockGrid, threadgrid, 0, stream[(curstream++)%n_streams]>>>(mu, BIG, n_elem);


  f32 *RE= lpar->RE, *CE= lpar->CE;
  n_elem = n_factors*4;
  dim3 blockGrid(n_elem/128 + 1);
  dim3 threadGrid(128);
  hipMemcpy(RE, V_data, n_elem * sizeof(f32), cudaDevicetoDevice);
  hipMemcpy(CE, V_data + n_elem * sizeof(f32), n_elem * sizeof(f32), cudaDevicetoDevice);
  gpu_multiply<<<blockGrid, threadgrid, 0, stream[(curstream++)%n_streams]>>>(RE, n_elem);
  gpu_multiply<<<blockGrid, threadgrid, 0, stream[(curstream++)%n_streams]>>>(CE, n_elem);

  for (i=0;i<n_streams;i++) {
    hipStreamSynchronize(stream[i]);
    hipStreamDestroy(stream[i]);
  }
  
  i32 *converged;
  hipMalloc(&converged,sizeof(i32));
  i32 _converged = 1;
  hipMemcpy(converged, &_converged, sizeof(i32), cudaHostToDevice);

  targs.com=com;
  targs.rom=rom;
  targs.co_pairs = co_pairs;
  targs.X=X;
  targs.refimg=NULL;
  targs.lpar = lpar;
  targs.self = self;
  targs.F_V = F_V;
  targs.V_F = V_F;
  targs.RE = RE;
  targs.CE = CE;
  targs.mu = mu;
  targs.unary_w=unary_w;
  targs.unary_c=unary_c;
  targs.X_py = X_py;
  targs.converged = converged;
  targs._converged = &_converged;


  for (it = 0; it < max_it; it++){
    if (it%10==0){
      printf("gpu it %d\n", it);
    }
    gpu_loopy_F_V(&targs);
    gpu_loopy_V_F(&targs);
    
    if (_converged) break;
  }
  printf("converged %d %f\n",_converged, lpar->stop_thresh);

  i32 *EY = lpar->EY;

  
  dim3 dimGrid(dims[0],dims[1]);
  dim3 singGrid(2);
  gpu_loopy_V_F__label<<<dimGrid, singGrid, 2*sizeof(f32)>>>(F_V, unary_c, EY, n_factors);

  //hipMemcpy(lpar->EY, EY, dims[0]*dims[1]*sizeof(i32), hipMemcpyDeviceToHost);
  
  
  
 cleanup:
  _mm_free(_F_V);
  _mm_free(_V_F);
  free(_com);
  free(_rom);
  free(_co_pairs);
  _mm_free(_RE);
  _mm_free(_CE);

  hipFree(F_V);
  hipFree(V_F);
  hipFree(com);
  hipFree(rom);
  hipFree(co_pairs);
  hipFree(RE);
  hipFree(CE);
  hipFree(unary_w);
  hipFree(unary_c);
  hipFree(mu);
  hipFree(X);
  hipFree(converged);
  hipFree(EY);

  return lpar->EY;
}


extern "C" void gpu_loopy_F_V(loopygpu_t *targs) { 

  npy_intp * dims= PyArray_DIMS(targs->X_py);
  gridCRF_t *self = targs->self;
  
  i32 n_factors=self->n_factors;
  
  f32 *F_V= targs->F_V;
  f32 *V_F= targs->V_F;

  f32 *RE = targs->RE;
  f32 *CE = targs->CE;

  i32 *com = targs->com;
  i32 *rom = targs->rom;
  om_pair * co_pairs = targs->co_pairs;
  
  i32 i;

  const i32 n_streams = 2;
  hipStream_t stream[n_streams];
  for (i=0;i<n_streams;i++) {
    hipStreamCreate(&stream[i]);
  }
  

  dim3 dimGrid(dims[0],dims[1]);
  dim3 factorgrid(n_factors);
		  
  gpu_loopy_F_V__Flow<<<dimGrid, factorgrid, 0, stream[0]>>>(F_V, V_F, RE, NULL, rom, co_pairs, n_factors);
  gpu_loopy_F_V__Fup<<<dimGrid, factorgrid, 0, stream[1]>>>(F_V, V_F, CE, NULL, com, co_pairs, n_factors);

  for (i=0;i<2;i++) {
    hipStreamDestroy(stream[i]);
  }

}


/* Naive method */
__global__ void gpu_loopy_F_V__Flow(f32 *F_V, f32 *V_F, f32 *RE, const i32 * refimg, const i32 * com, const om_pair * co_pairs,  i32 n_factors){
  /* Naive code*/
  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 n = threadIdx.x;
  //Note, may need to swap gridDim.x and gridDim.y

  /* Check bounds for upper factor */
  om_pair cop = co_pairs[n];
  if ( ! (x+cop.x <0 || x+cop.x >= gridDim.x || y+cop.y < 0 || y+cop.y >=gridDim.y) ){//&& !(refimg[COORD2(x+cop.x,y+cop.y, gridDim.x, gridDim.y, 1)]==0)) {
    i32 origin=COORD3(x,y,0,gridDim.x,gridDim.y,2*n_factors,2);
    i32 co = origin + com[n] + 2*(n + n_factors);
  
    F_V[co] = RE[n*2] + V_F[origin] > RE[n_factors*2 + n*2] + V_F[origin+1] ? RE[n*2] + V_F[origin] : RE[n_factors*2 + n*2] + V_F[origin+1];
  
    F_V[co+1] = RE[n*2+1] + V_F[origin] > RE[n_factors*2 + n*2 + 1] + V_F[origin+1] ? RE[n*2+1] + V_F[origin] : RE[n_factors*2 + n*2 + 1] + V_F[origin+1];
  }
}

__global__ void gpu_loopy_F_V__Fup(f32 *F_V, f32 *V_F,  f32 *CE, const i32 * refimg, const i32 *rom, const om_pair *co_pairs, i32 n_factors){
  /* Naive code*/
  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 n = threadIdx.x;
  

  /* Check bounds for upper factor */
  om_pair cop=co_pairs[n];
  /* Check bounds for lower factor */
  if (!(x-cop.x < 0 || x-cop.x >= gridDim.x || y-cop.y < 0 || y-cop.y >=gridDim.y)){// && !(refimg[COORD2(x-cop.x,y-cop.y, gridDim.x, gridDim.y, 1)]==0)) {
    i32 origin=COORD3(x,y,0,gridDim.x,gridDim.y,2*n_factors,2);
    //i32 co=origin + rom[n]; //check this
    i32 co = origin+rom[n] + 2*n;
    F_V[co] = CE[n*2] + V_F[origin] > CE[n_factors*2 + n*2] + V_F[origin+1] ?
      CE[n*2] + V_F[origin] : CE[n_factors*2 + n*2] + V_F[origin+1];
  
    F_V[co+1] = CE[n*2+1] + V_F[origin] > CE[n_factors*2 + n*2 + 1] + V_F[origin+1] ?
      CE[n*2+1] + V_F[origin] : CE[n_factors*2 + n*2 + 1] + V_F[origin+1];
  }
  
}



extern "C" void gpu_loopy_V_F(loopygpu_t *targs) {


  //TODO:
  // - copy X to gpu

  i32 i;
  gridCRF_t *self = targs->self;
  f32 *X = targs->X;

  loopy_params_t * lpar = targs->lpar;
  
  npy_intp * dims= PyArray_DIMS(targs->X_py);
  i64 n_factors=self->n_factors;
  f32 stop_thresh=lpar->stop_thresh;
  

  f32 * unary_w= targs->unary_w;
  f32 * unary_c = targs->unary_c;

  f32 *F_V = targs->F_V;
  f32 *V_F = targs->V_F;
  f32 *mu = targs->mu;

  /* runtime Flags*/
  i32 *converged = targs->converged;
  
  const i32 n_streams = 1;
  hipStream_t stream[n_streams];
  for (i=0;i<n_streams;i++) {
    hipStreamCreate(&stream[i]);
  }
  
  i32 tmp=1;
  hipMemcpyAsync(converged, &tmp, sizeof(i32),hipMemcpyHostToDevice,stream[0]);

  for (i=0;i<n_streams;i++) {
    hipStreamSynchronize(stream[i]);
  }

  
  dim3 dimGrid(dims[0],dims[1]);
  dim3 factorgrid(2*n_factors,2);
  dim3 singGrid(2);
  gpu_loopy_V_F__computeunary<<<dimGrid, singGrid, 0, stream[0]>>>(X, unary_w, unary_c);
  gpu_loopy_V_F__sumfactors<<<dimGrid, factorgrid, sizeof(f32)*n_factors*8, stream[0]>>>(F_V, V_F, unary_c, NULL, n_factors);
  gpu_loopy_V_F__marginal<<<dimGrid, singGrid, 0, stream[0]>>>(F_V, unary_c, mu, n_factors, stop_thresh, converged);

  for (i=0;i<n_streams;i++) {
    hipStreamDestroy(stream[i]);
  }


  hipMemcpy(targs->_converged, converged, sizeof(i32), hipMemcpyDeviceToHost);
 
}

__global__ void gpu_loopy_V_F__computeunary(f32 * X, f32 *unary_w, f32 *unary_c){
    // one possibility. X is not aligned properly.
  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 c = threadIdx.x;

  unary_c[COORD2(x,y,gridDim.x, gridDim.y, 2) + c] = -(	    \
    X[COORD2(x,y,gridDim.x, gridDim.y, 2)] * unary_w[c*2] + \
      X[COORD2(x,y,gridDim.x, gridDim.y, 2) + 1] * unary_w[c*2 + 1]);
    
}

__global__ void gpu_loopy_V_F__sumfactors(f32 *F_V, f32 *V_F, f32 *unary_c, const i32 * refimg, i32 n_factors ){
  extern __shared__ char array[];
  f32 *shared_f_v = (f32*) array;
  f32 *shared_v_f = (f32*) (array + sizeof(f32)*n_factors*2*2);

  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 n = threadIdx.x;
  i32 c = threadIdx.y;
  i32 i=0;
  i32 origin = COORD3(x,y,n,gridDim.x, gridDim.y, 2*n_factors, 2) + c;
  // load factor to vvariables into shared memory
  shared_f_v[n*2 + c] = F_V[origin];

  //TODO: make unary a constant?
  f32 sum = unary_c[COORD2(x,y,gridDim.x,gridDim.y,2) + c] - shared_f_v[n*2+c];
  __syncthreads();


  /* Sum up all messages */
  for (i=0;i<2*n_factors;i++) {
//printf("KEK %d\n", i*2+c);
    sum += shared_f_v[i*2+c]; //this is the problem!
  }
  shared_v_f[n*2+c]=sum;
  
  __syncthreads();  
  // Normalize values
  sum = sum - 0.5 * (sum+shared_v_f[n*2+c^1]);
  V_F[origin]=sum;
  
}

/* TODO: finish */
__global__ void gpu_loopy_V_F__marginal(f32 *F_V, f32 * unary_c,  f32 * mu, i32 n_factors, f32 stop_thresh, i32 *converged) {
  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 c = threadIdx.x;

  i32 i;
  i32 origin = COORD2(x,y,gridDim.x, gridDim.y, 2) + c;
  f32 sum = unary_c[origin];

  // sum up factors
  for (i=0;i<n_factors*2;i++) {
    sum += F_V[COORD3(x,y,i,gridDim.x, gridDim.y, 2*n_factors, 2) + c];
  }

  if (fabsf(sum - mu[origin]) > stop_thresh) {
    converged[0] = 0;
  }
  mu[origin] = sum;
}

//TODO: change to mu instead
__global__ void gpu_loopy_V_F__label(f32 *F_V, f32 * unary_c, i32 *EY, i32 n_factors) {

  /* Computes the predicted label given the values */
  extern __shared__ char array[];
  f32 *shared_marginal = (f32*) array;
  i32 i;
  i32 x = blockIdx.x;
  i32 y = blockIdx.y;
  i32 c = threadIdx.x;
  i32 origin = COORD2(x,y,gridDim.x, gridDim.y, 2) + c;

  f32 sum = unary_c[origin];
  // sum up factors
  for (i=0;i<n_factors*2;i++) {
    sum += F_V[COORD3(x,y,i,gridDim.x, gridDim.y, 2*n_factors, 2) + c];
  }
  shared_marginal[c] = sum;
  __syncthreads();
  if (c==0  && shared_marginal[0] > shared_marginal[1]) {
    EY[COORD2(x,y,gridDim.x, gridDim.y, 1)] = 0;
  }
  else if (c==0) {
    EY[COORD2(x,y,gridDim.x, gridDim.y, 1)] = 1;
  }
  
}


__global__ void gpu_fill_value(f32 *buffer, f32 val, i32 lim) {
  i32 index = blockIdx * blockDim.x + threadIdx.x;
  if (index < lim) {buffer[index] = val;}
}

__global__ void gpu_multiply(f32 *buffer, i32 lim) {
   i32 index = blockIdx * blockDim.x + threadIdx.x;
   if (index < lim) {buffer[index] = -buffer[index];}
}


#ifdef CPU_TEST



void __debug_loopy_V_F(loopygpu_t *targs) {

  /* Compute factor to variable messages */
  i32 WARN_FLAG=1;
  i32 i,j;
  gridCRF_t *self = targs->self;
  PyArrayObject *X = targs->X_py;
  loopy_params_t * lpar = targs->lpar;
  
  f32 a,b;
  npy_intp * dims= PyArray_DIMS(X);
  i64 n_factors=self->n_factors;
  f32 stop_thresh=lpar->stop_thresh;
  
  f32 * unary = self->unary;
  
  npy_intp x,y;
  i32 n;


  f32 *F_V = targs->_F_V;
  f32 *V_F = targs->_V_F;

  f32 *gF_V = targs->F_V;
  f32 *gV_F = targs->V_F;

  hipMemcpy(F_V, gF_V, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32), hipMemcpyDeviceToHost);
  hipMemcpy(V_F, gV_F, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32), hipMemcpyDeviceToHost);
  
  
  f32 *marginals = (f32*) _mm_malloc(sizeof(f32)*dims[0]*dims[1]*2,32);
  f32 *mu = targs->_mu;

  /* coordinates */

  /* runtime Flags*/
  i32 *converged = targs->_converged;
  
  i32 origin;

  f32 tmp[2];

  __m256 r1,r2;
  /* Compute variable to factor messages */
  for (x=0;x<dims[0];x++) {
    for (y=0;y<dims[1];y++) {
      //variable to factor messages
      
      f64 base= *((f64*)PyArray_GETPTR3(X,x,y,0));
      *((f64*)tmp) = base;
      tmp[0]=-(((f32*)&base)[0]*unary[0] + ((f32*)&base)[1]*unary[1]);
      tmp[1]=-(((f32*)&base)[0]*unary[2] + ((f32*)&base)[1]*unary[3]);
      r1=(__m256)_mm256_set1_pd(*((f64*)tmp)); //set all elements in vector this thi
      //Warning: possible segfault
	
      for (n=0;n<n_factors*2;n+=4) { //Set baseline, since we know that unary is added to each V_F
	_mm256_store_ps(&V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)] ,r1);
      }
      
      for (i=0;i<n_factors*2;i++) {
	base=*((f64*)(&F_V[COORD3(x,y,i,dims[0],dims[1],2*n_factors,2)]));
	r1=(__m256)_mm256_set1_pd(base);
	for (n=0;n<n_factors*2;n+=4) {
	  r2=_mm256_load_ps(&V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)]);
	  r2=_mm256_add_ps(r2,r1);
	  _mm256_store_ps(&V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)],r2);
	}
      }
      for (n=0;n<n_factors*2;n+=8) { //correct double counting
	r1=_mm256_load_ps(&F_V[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)]);
	r2=_mm256_load_ps(&V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)]);
	r2=_mm256_sub_ps(r2,r1);
	_mm256_store_ps(&V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)],r2);
      }

      /*
      //TODO: normalize
      //This is SSE normalization. Unless more knowledge is gained, it would be slower to use these than individually going over each value.
      */
	
      //Apply normalization
      for (n=0;n<n_factors*2;n++) {
	//TODO: optimize
	a=V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)];
	b=V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)+1];
	a=0.5*(a+b);
	V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)]-=a;
	V_F[COORD3(x,y,n,dims[0],dims[1],2*n_factors,2)+1]-=a;
      }
	
      //TODO:Add to marginals
      origin=COORD2(x,y,dims[0],dims[1],2);

      assert (origin < dims[0]*dims[1]*2);

      marginals[origin]=tmp[0];
      marginals[origin+1]=tmp[1];
      for (i=0;i<n_factors*2;i++) {

	assert(COORD3(x,y,i,dims[0],dims[1],2*n_factors,2)+ 1 < dims[0] * dims[1] * (n_factors*2) *2 && COORD3(x,y,n,dims[0],dims[1],2*n_factors,2) > 0);
	marginals[origin]+=F_V[COORD3(x,y,i,dims[0],dims[1],2*n_factors,2)]; // invalid read of 4
	marginals[origin+1]+=F_V[COORD3(x,y,i,dims[0],dims[1],2*n_factors,2)+1];
	if (isnan(marginals[origin]) && WARN_FLAG) {
	  printf("MARGINAL WARNING %d %d\n",x,y);
	  for (j=0;j<n_factors*2;j++) {
	    printf("%f %f\n",F_V[COORD3(x,y,j,dims[0],dims[1],2*n_factors,2)],F_V[COORD3(x,y,j,dims[0],dims[1],2*n_factors,2)+1]); // invalid read of 4
	  }
	  WARN_FLAG=0;
	}
      }
      a=fabs(marginals[origin]-mu[origin]);
      if (a > stop_thresh) {
	*converged = 0;
      }
      a=fabs(marginals[origin+1]-mu[origin+1]);
      if (a > stop_thresh) {
	*converged = 0;
      }

      mu[origin]=marginals[origin];
      mu[origin+1]=marginals[origin+1];
      //TODO: calculate marginal
    }
  }

  _mm_free(marginals);
  hipMemcpy(gF_V, F_V, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32), hipMemcpyHostToDevice);
  hipMemcpy(gV_F, V_F, dims[0] * dims[1] * (n_factors*2) *2* sizeof(f32), hipMemcpyHostToDevice);
}


void *_loopy_label(loopygpu_t *l_args) {
  loopy_params_t * lpar = l_args->lpar;
  npy_intp * dims= PyArray_DIMS(l_args->X_py);
  i32 *ret=lpar->EY;
  f32 *mu=l_args->_mu;
  i32 x,y;
  i32 origin;
  for (x=0;x<dims[0];x++) {
    for (y=0;y<dims[1];y++) {
   
      origin=COORD2(x,y,dims[0],dims[1],2); 
      assert(origin >= 0 && origin + 1 < dims[0]*dims[1]*2);
      if (mu[origin] > mu[origin+1]) {
	ret[COORD2(x,y,dims[0],dims[1],1)]=0;
      }
      else{
	ret[COORD2(x,y,dims[0],dims[1],1)]=1;

      }
    }

  }

  //hipMemcpy(ret, gEY, dims[0] * dims[1] * sizeof(i32), hipMemcpyHostToDevice);

  return NULL;

}
#endif
